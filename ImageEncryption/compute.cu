#include "hip/hip_runtime.h"
#include "compute.cuh"
#include<iostream>
using namespace std;
void imgRowExchange(Mat& img, int* p)
{
    for (int i = 0; i < img.rows; i++) {
        Mat temp1 = img.row(i).clone();
        Mat temp2 = img.row(p[i]).clone();

        temp1.copyTo(img.row(p[i]));
        temp2.copyTo(img.row(i));
    }
}

void imgColExchange(Mat& img, int* q)
{
    for (int j = 0; j < img.cols; j++) {
        Mat temp1 = img.col(j).clone();
        Mat temp2 = img.col(q[j]).clone();
        temp1.copyTo(img.col(q[j]));
        temp2.copyTo(img.col(j));
    }
}

void imgConfusion(Mat& img)
{
    //获取2D-HSM混沌序列
    int n = img.rows+img.cols;
    double* temp_X = new double[n], * temp_Y = new double[n];
    int* row_X = new int[n], * row_Y = new int[n];
    getSine2DSequence(temp_X, temp_Y, n);

    for (int i = 0; i < n; i++) {
        row_X[i] = static_cast<long long>(temp_X[i] * 1e14)% img.rows;
        row_Y[i] = static_cast<long long>(temp_Y[i] * 1e14)% img.rows;
        if (row_X[i] < 0)  row_X[i] = -row_X[i];
        if (row_Y[i] < 0)  row_Y[i] = -row_Y[i];
    }

    int* col_X = new int[n], * col_Y = new int[n];
    getSine2DSequence(temp_X, temp_Y, n);
    for (int i = 0; i < n; i++) {
        col_X[i] = static_cast<long long>(temp_X[i] * 1e14) % img.cols;
        col_Y[i] = static_cast<long long>(temp_Y[i] * 1e14) % img.cols;

        if (col_X[i] < 0)  col_X[i] = -col_X[i];
        if (col_Y[i] < 0)  col_Y[i] = -col_Y[i];
    }
    delete[] temp_X;
    delete[] temp_Y;

    for(int i=0;i<n;i++){
        //交换行
        int index=i%img.rows;
        Mat temp1 = img.row(index).clone();
        Mat temp2 = img.row(row_X[i]).clone();

        temp1.copyTo(img.row(row_X[i]));
        temp2.copyTo(img.row(index));

        //交换列
        index=i%img.cols;
        Mat temp3 = img.col(index).clone();
        Mat temp4 = img.col(col_X[i]).clone();
        temp3.copyTo(img.col(col_X[i]));
        temp4.copyTo(img.col(index));
    }

    for(int i=0;i<n;i++){
        //交换行
        int index=i%img.rows;
        Mat temp1 = img.row(index).clone();
        Mat temp2 = img.row(row_Y[i]).clone();

        temp1.copyTo(img.row(row_Y[i]));
        temp2.copyTo(img.row(index));

        //交换列
        index=i%img.cols;
        Mat temp3 = img.col(index).clone();
        Mat temp4 = img.col(col_Y[i]).clone();
        temp3.copyTo(img.col(col_Y[i]));
        temp4.copyTo(img.col(index));
    }

    delete[] row_X;
    delete[] row_Y;
    delete[] col_X;
    delete[] col_Y;
}

void chaosSelectByCol(int* chaos, double* update, int n)
{
    getLogisticSequence(update, n);
    for (int i = 0; i < n; i++) {
        chaos[i] = static_cast<long long>(update[i] * 1e14) % 3;
    }
}

void updateKeys(unsigned char buf[])
{

    //更新密钥
    double CK = std_a + std_b+std_k + cat3D_x + cat3D_y + cat3D_z + lorenz_y + lorenz_z + lorenz_q + sine2D_a + sine2D_b + sine2D_x + sine2D_y + logistic_x;
    CK = fmod(CK, 1);

    std_a = std_a + ((static_cast<long long>(buf[0]) << 8) + buf[1]) / 65536.0 + CK; std_a = fmod(std_a, 2*pi);
    std_b = std_b + ((static_cast<long long>(buf[2]) << 8) + buf[3] ) / 65536.0 + CK; std_b = fmod(std_b, 2*pi);
    std_k = std_k + ((static_cast<long long>(buf[4]) << 8) + buf[5]) / 65536.0 + CK; std_k =fmod(std_k, 1);

    cat3D_x = cat3D_x + ((static_cast<long long>(buf[6]) << 8) + buf[7] ) / 65536.0 + CK; cat3D_x =fmod(cat3D_x, 1);
    cat3D_y = cat3D_y + ((static_cast<long long>(buf[8]) << 8) + buf[9] ) / 65536.0 + CK; cat3D_y= fmod(cat3D_y, 1);
    cat3D_z = cat3D_z + ((static_cast<long long>(buf[10]) << 8) + buf[11] ) / 65536.0 + CK; cat3D_z=fmod(cat3D_z, 1);

    lorenz_y = lorenz_y + ((static_cast<long long>(buf[12]) << 8) + buf[13] ) / 65536.0 + CK; lorenz_y=fmod(lorenz_y, 1);
    lorenz_z = lorenz_z + ((static_cast<long long>(buf[14]) << 8) + buf[15] ) / 65536.0 + CK; lorenz_z=fmod(lorenz_z, 1);
    lorenz_q = lorenz_q + ((static_cast<long long>(buf[16]) << 8) + buf[17] ) / 65536.0 + CK; lorenz_q=fmod(lorenz_q, 1);

    logistic_x = logistic_x + ((static_cast<long long>(buf[18]) << 8) + buf[19]) / 65536.0 + CK; logistic_x = fmod(logistic_x, 1);

    sine2D_a = sine2D_a + ((static_cast<long long>(buf[20]) << 8)+(static_cast<long long>(buf[21]) << 8)
        + buf[22] ) / 65536.0/256.0 + CK; 
    sine2D_b = sine2D_b + ((static_cast<long long>(buf[23]) << 8) + (static_cast<long long>(buf[24]) << 8)
        + buf[25]) / 65536.0 / 256.0 + CK;
    sine2D_x = sine2D_x + ((static_cast<long long>(buf[26]) << 8) + (static_cast<long long>(buf[27]) << 8)
        + buf[28]) / 65536.0 / 256.0 + CK;
    sine2D_x=fmod(sine2D_x, 1);
    sine2D_y = sine2D_y + ((static_cast<long long>(buf[29]) << 8) + (static_cast<long long>(buf[30]) << 8)
        + buf[31]) / 65536.0 / 256.0 + CK;
    sine2D_y = fmod(sine2D_y, 1);
    /*
    cout << sine_x << "--" << sine_u << "--" << cat3D_x << "--" <<cat3D_y << "--" << cat3D_z
        << "--" << lorenz_y << "--" << lorenz_z << "--" << lorenz_q << "--" << sine2D_a << "--" << sine2D_b
        << "--" << sine2D_x << "--" << sine2D_y << "--" <<logistic_u << "--" << logistic_x<<endl;
    */

    hipMemcpyToSymbol(HIP_SYMBOL(dev_std_a), &std_a, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_std_b), &std_b, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_std_k), &std_k, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_cat3D_x), &cat3D_x, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_cat3D_y), &cat3D_y, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_cat3D_z), &cat3D_z, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_lorenz_y), &lorenz_y, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_lorenz_z), &lorenz_z, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_lorenz_q), &lorenz_q, sizeof(double));

}

void deImgConfusion(Mat& img)
{
    //获取2D-HSM混沌序列
    int n = img.rows+img.cols;
    double* temp_X = new double[n], * temp_Y = new double[n];
    int* row_X = new int[n], * row_Y = new int[n];
    getSine2DSequence(temp_X, temp_Y, n);

    for (int i = 0; i < n; i++) {
        row_X[i] = static_cast<long long>(temp_X[i] * 1e14)% img.rows;
        row_Y[i] = static_cast<long long>(temp_Y[i] * 1e14)% img.rows;
        if (row_X[i] < 0)  row_X[i] = -row_X[i];
        if (row_Y[i] < 0)  row_Y[i] = -row_Y[i];
    }

    int* col_X = new int[n], * col_Y = new int[n];
    getSine2DSequence(temp_X, temp_Y, n);
    for (int i = 0; i < n; i++) {
        col_X[i] = static_cast<long long>(temp_X[i] * 1e14) % img.cols;
        col_Y[i] = static_cast<long long>(temp_Y[i] * 1e14) % img.cols;

        if (col_X[i] < 0)  col_X[i] = -col_X[i];
        if (col_Y[i] < 0)  col_Y[i] = -col_Y[i];
    }
    delete[] temp_X;
    delete[] temp_Y;

    for(int i=n-1;i>=0;i--){
        //交换列
        int index=i%img.cols;
        Mat temp3 = img.col(index).clone();
        Mat temp4 = img.col(col_Y[i]).clone();
        temp3.copyTo(img.col(col_Y[i]));
        temp4.copyTo(img.col(index));


        //交换行
        index=i%img.rows;
        Mat temp1 = img.row(index).clone();
        Mat temp2 = img.row(row_Y[i]).clone();

        temp1.copyTo(img.row(row_Y[i]));
        temp2.copyTo(img.row(index));

    }


    for(int i=n-1;i>=0;i--){
        //交换列
        int index=i%img.cols;
        Mat temp3 = img.col(index).clone();
        Mat temp4 = img.col(col_X[i]).clone();
        temp3.copyTo(img.col(col_X[i]));
        temp4.copyTo(img.col(index));

        //交换行
        index=i%img.rows;
        Mat temp1 = img.row(index).clone();
        Mat temp2 = img.row(row_X[i]).clone();

        temp1.copyTo(img.row(row_X[i]));
        temp2.copyTo(img.row(index));
    }

    delete[] row_X;
    delete[] row_Y;
    delete[] col_X;
    delete[] col_Y;
}

void deImgColExchange(Mat &img, int *q) {
    for (int j = img.cols-1; j >=0; j--) {
        Mat temp1 = img.col(j).clone();
        Mat temp2 = img.col(q[j]).clone();
        temp1.copyTo(img.col(q[j]));
        temp2.copyTo(img.col(j));
    }
}
void deImgRowExchange(Mat& img, int* p)
{
    for (int i = img.rows-1; i >=0; i--) {
        Mat temp1 = img.row(i).clone();
        Mat temp2 = img.row(p[i]).clone();

        temp1.copyTo(img.row(p[i]));
        temp2.copyTo(img.row(i));
    }
}