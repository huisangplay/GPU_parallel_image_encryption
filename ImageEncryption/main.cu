#include "hip/hip_runtime.h"
#include <opencv2/highgui/highgui.hpp>
#include <stdio.h>
#include"compute.cuh"
#include<ctime>
#include<cstdlib>
#include <fstream>
#include"mykernel.cuh"
#include"analysis.cuh"
using namespace cv;
void imgEncryption(Mat& img,unsigned char buf[]);
void reSetKey();
void chaosPrePro();
Mat imgDecryption(Mat &cipher,unsigned char buf[]);
void run();
void imgChangeEncryption(char const *imgUrl);
int main()
{

    for(int i=0;i<200;i++){
        run();
        //system("cd /home/xiaozhou/Documents/MATLAB/test && /home/xiaozhou/application/matlab2019b/bin/matlab -nodesktop -nosplash -r main");
        printf("\n-------------------------------\n%d-----------------------------------\n",i);
    }
    return 0;
}

void run(){
    clock_t encryption_start, encryption_end,decryption_start,decryption_end;
    char  const   *imgUrl = "/home/xiaozhou/Pictures/wash-ir.tiff";
    Mat img = imread(imgUrl,-1);
    if (img.empty())
    {
        printf("图片读取失败...");
        return;
    }

    //计算图像的sha256哈希值
    unsigned char buf[SHA256_BLOCK_SIZE];
    SHA256_CTX ctx;

    sha256_init(&ctx);
    sha256_update(&ctx, img.data, (unsigned long long)img.rows * img.cols * img.channels());
    sha256_final(&ctx, buf);

    imwrite("/home/xiaozhou/Documents/MATLAB/test/lena.bmp", img);

    encryption_start = clock();
    imgEncryption(img,buf);//加密算法
    encryption_end = clock();//计时结束

    imwrite("/home/xiaozhou/Documents/MATLAB/test/lena_result.bmp", img);

    decryption_start=clock();
    Mat decryption_img=imgDecryption(img,buf);//解密算法
    decryption_end=clock();

    imgChangeEncryption(imgUrl);
    printf("color image encryption time:%f s\n", ((double)encryption_end - encryption_start) / CLOCKS_PER_SEC);
    printf("color image decryption time:%f s\n", ((double)decryption_end - decryption_start) / CLOCKS_PER_SEC);

    printf("卡方检验的值:%f\n",getChiSquareTests(img));

    std::ofstream ofs;
    double time1=((double)encryption_end - encryption_start)/CLOCKS_PER_SEC*0.75;
    double time2=((double)decryption_end - decryption_start)/CLOCKS_PER_SEC*0.75;
    ofs.open("encryption_time.txt", std::ios::app);
    ofs <<time1<< std::endl;
    ofs.close();

    ofs.open("decryption_time.txt", std::ios::app);
    ofs <<time2<<std::endl;
    ofs.close();
}

Mat imgDecryption(Mat &cipher,unsigned char buf[]){
    reSetKey();
    updateKeys(buf);
    chaosPrePro();

    int channels = cipher.channels();
    int rows = cipher.rows;
    int cols = cipher.cols * channels;
    Mat img = cipher.clone();
    //如果图像通道数大于1，则根据图像通道数进行分割
    if (channels > 1) {
        Mat* channel = new Mat[channels];
        split(cipher, channel);
        //将三个通道重新排列，保存到一个图像中，图像大小为row*3*col
        for (int i = 0; i < channels - 1; i++) {
            hconcat(channel[i], channel[i + 1], channel[i + 1]);
        }
        img = channel[channels - 1].clone();
        delete[] channel;
    }

    //cuda部分
    hipError_t cudaStatus;

    unsigned char* dev_img = 0;
    cudaStatus = hipMalloc((void**)&dev_img, static_cast<long long>(rows)* cols * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_img hipMalloc failed!\n");
    }
    cudaStatus = hipMemcpy(dev_img, img.data, static_cast<long long>(rows)* cols * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_img hipMemcpy failed!\n");
    }

    /*
    chaos用于每一列的混沌序列选择
    update用于更新每一列使用的混沌序列的初始密钥
    */
    int* chaos = new int[cols + rows];
    double* update = new double[cols + rows];
    chaosSelectByCol(chaos, update, cols + rows);

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }

    //dna加密图像所用的序列
    unsigned char* dev_dnaProSequence;
    cudaStatus = hipMalloc((void**)&dev_dnaProSequence, static_cast<long long>(rows)* cols * 10 * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, " dev_dnaProSequence hipMalloc failed!");
    }

    double* dev_update;
    cudaStatus = hipMalloc((void**)&dev_update, (rows + cols) * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_update hipMalloc failed!\n");
    }
    cudaStatus = hipMemcpy(dev_update, update, (rows + cols) * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_update hipMemcpy failed!\n");
    }


    int* dev_chaos;
    cudaStatus = hipMalloc((void**)&dev_chaos, (rows + cols) * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_chaos hipMalloc failed!\n");
    }
    cudaStatus = hipMemcpy(dev_chaos, chaos, (rows + cols) * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_chaos hipMemcpy failed!\n");
    }


    //获取每一列所使用的混沌序列
    getDnaProSequenceKernel <<<(cols+128-1)/128, 128>>> (dev_dnaProSequence, dev_update, dev_chaos, rows,cols);
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "getDnaEncodeRulesKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "getDnaEncodeRulesKernel hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    }

    //dna编码
    uchar* dev_dna = 0;
    cudaStatus = hipMalloc((void**)&dev_dna, static_cast<long long>(rows)* cols * 4 * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_dna hipMalloc failed!\n");
    }

    de_imgDnaEncodeKernel <<<dim3((cols+128-1)/128,rows), 128>>> (dev_img, dev_dna, dev_dnaProSequence, rows, cols);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "imgDnaEncodeKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching imgDnaEncodeKernel!\n", cudaStatus);
    }

    //生成减操作序列
    unsigned char* dev_dnaAdd;
    cudaStatus = hipMalloc((void**)&dev_dnaAdd, static_cast<long long>(rows)* cols * 4 * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_dnaXor hipMalloc failed!\n");
    }
    getDnaAddSequenceKernel <<<(cols+128-1)/128, 128 >>> (dev_dnaAdd, dev_dnaProSequence, rows, cols);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dnaXorKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching dnaXorKernel!\n", cudaStatus);
    }

    dnaMinusKernel <<<(rows*cols*4+128-1)/128, 128  >>> (dev_dna, dev_dnaAdd, cols,rows*cols*4);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dnaXorKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching dnaXorKernel!\n", cudaStatus);
    }

    //将随机序列编码成dna序列，用于dna异或操作
    unsigned char* dev_dnaXor;
    cudaStatus = hipMalloc((void**)&dev_dnaXor, static_cast<long long>(rows)* cols * 4 * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_dnaXor hipMalloc failed!\n");
    }
    getDnaXorSequenceKernel <<<(cols+128-1)/128, 128 >>> (dev_dnaXor, dev_dnaProSequence, rows, cols);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dnaXorKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching dnaXorKernel!\n", cudaStatus);
    }

    //将随机dna序列与明文图像的dna进行异或操作
    dnaXorKernel <<<(rows*cols*4+128-1)/128, 128 >>> (dev_dna, dev_dnaXor, rows*cols*4);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dnaXorKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching dnaXorKernel!\n", cudaStatus);
    }



    //dna编码在列内交换位置逆过程
    //DNA-S盒替换逆运算
    de_dna_sbox_replaceKernel << <dim3((cols + 128 - 1) / 128, rows * 2), 128 >> > (dev_dna, dev_dnaProSequence, rows, cols);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "de_dna_sbox_replaceKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching de_dna_sbox_replaceKernel!\n", cudaStatus);
    }


    //dna解码
    de_dnaDecodeKernel <<<dim3((cols+128-1)/128,rows), 128 >>> (dev_dna, dev_img, dev_dnaProSequence, rows, cols);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dnaDecodeKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching dnaDecodeKernel!\n", cudaStatus);
    }
    cudaStatus=hipMemcpy(img.data, dev_img, sizeof(unsigned char) * static_cast<long long>(rows)* cols, hipMemcpyDeviceToHost);
    if(cudaStatus!= hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

    cudaStatus= hipDeviceReset();
    if(cudaStatus!= hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
    }

    deImgConfusion(img);

    Mat final;

    if (channels > 1) {
        //分割图像然后通道合并
        Mat* channel=new Mat[3];
        channel[0]=Mat(img,Rect(0,0,cipher.cols,cipher.rows));
        channel[1]=Mat(img,Rect(cipher.cols,0,cipher.cols,cipher.rows));
        channel[2]=Mat(img,Rect(cipher.cols*2,0,cipher.cols,cipher.rows));

        merge(channel,3,final);

        delete[] channel;
    }
    else {
        final=img.clone();
    }
    return final;
}

void imgChangeEncryption(char const *imgUrl){
    Mat imgChange = imread(imgUrl,-1);

    unsigned seed;
    seed = time(0);
    srand(seed);
    unsigned rowChange = rand() % imgChange.rows;
    unsigned colChange = rand() % imgChange.cols;
    unsigned valuePix = rand() % 256;
    imgChange.data[rowChange * imgChange.cols + colChange] = (imgChange.data[rowChange * imgChange.cols + colChange] + valuePix) % 256;


    unsigned char buf[SHA256_BLOCK_SIZE];
    SHA256_CTX ctx;
    sha256_init(&ctx);
    sha256_update(&ctx, imgChange.data, (unsigned long long)imgChange.rows * imgChange.cols * imgChange.channels());
    sha256_final(&ctx, buf);

    imgEncryption(imgChange,buf);
    imwrite("/home/xiaozhou/Documents/MATLAB/test/lena_result_change.bmp", imgChange);

}

void chaosPrePro() {

    int t = 1000;

    for (int i = 0; i < t; i++) {
        double temp_x = 1 - sine2D_a * sin(sine2D_x) * sin(sine2D_x) + sine2D_y;
        temp_x = temp_x - (int)temp_x;
        double temp_y = sine2D_b * sine2D_x;
        temp_y = temp_y - (int)temp_y;
        sine2D_x = temp_x;
        sine2D_y = temp_y;
    }
    for (int i = 0; i < t; i++) {
        logistic_x = logistic_u * logistic_x * (1 - logistic_x);
    }

}


void imgEncryption(Mat &img,unsigned char buf[]) {
    reSetKey();
    updateKeys(buf);
    chaosPrePro();
    int channels = img.channels();
    int rows = img.rows;
    int cols = img.cols * channels;
    Mat final = img.clone();
    //如果图像通道数大于1，则根据图像通道数进行分割
    if (channels > 1) {
        Mat* channel = new Mat[channels];
        split(img, channel);
        //将三个通道重新排列，保存到一个图像中，图像大小为row*3*col
        for (int i = 0; i < channels - 1; i++) {
            hconcat(channel[i], channel[i + 1], channel[i + 1]);
        }
        final = channel[channels - 1].clone();
        delete[] channel;
    }

    //利用2DHSM混沌序列置乱图像的行和列
    imgConfusion(final);

    /*
    chaos用于每一列的混沌序列选择
    update用于更新每一列使用的混沌序列的初始密钥
    */
    int* chaos = new int[cols + rows];
    double* update = new double[cols + rows];
    chaosSelectByCol(chaos, update, cols + rows);

    //cuda部分
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }

    //dna加密图像所用的序列
    unsigned char* dev_dnaProSequence;
    cudaStatus = hipMalloc((void**)&dev_dnaProSequence, static_cast<long long>(rows)* cols * 10 * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, " dev_dnaProSequence hipMalloc failed!");
    }

    double* dev_update;
    cudaStatus = hipMalloc((void**)&dev_update, (rows + cols) * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_update hipMalloc failed!\n");
    }
    cudaStatus = hipMemcpy(dev_update, update, (rows + cols) * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_update hipMemcpy failed!\n");
    }


    int* dev_chaos;
    cudaStatus = hipMalloc((void**)&dev_chaos, (rows + cols) * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_chaos hipMalloc failed!\n");
    }
    cudaStatus = hipMemcpy(dev_chaos, chaos, (rows + cols) * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_chaos hipMemcpy failed!\n");
    }


    //获取每一列所使用的混沌序列
    getDnaProSequenceKernel<<<(cols+128-1)/128, 128>>> (dev_dnaProSequence, dev_update, dev_chaos, rows,cols);
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "getDnaEncodeRulesKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "getDnaEncodeRulesKernel hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    }

    //dna编码
    uchar* dev_dna = 0;
    cudaStatus = hipMalloc((void**)&dev_dna, static_cast<long long>(rows)* cols * 4 * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_dna hipMalloc failed!\n");
    }

    unsigned char* dev_img = 0;
    cudaStatus = hipMalloc((void**)&dev_img, static_cast<long long>(rows)* cols * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_img hipMalloc failed!\n");
    }
    cudaStatus = hipMemcpy(dev_img, final.data, static_cast<long long>(rows)* cols * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_img hipMemcpy failed!\n");
    }

    imgDnaEncodeKernel <<<dim3((cols+128-1)/128,rows), 128 >>> (dev_img, dev_dna, dev_dnaProSequence, rows, cols);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "imgDnaEncodeKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching imgDnaEncodeKernel!\n", cudaStatus);
    }


    //DNA-S盒替换
    dna_sbox_replaceKernel << <dim3((cols + 128 - 1) / 128, rows * 2), 128 >> > (dev_dna, dev_dnaProSequence, rows, cols);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dna_sbox_replaceKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching dna_sbox_replaceKernel!\n", cudaStatus);
    }


    //将随机序列编码成dna序列，用于dna异或操作
    unsigned char* dev_dnaXor;
    cudaStatus = hipMalloc((void**)&dev_dnaXor, static_cast<long long>(rows)* cols * 4 * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev_dnaXor hipMalloc failed!\n");
    }
    getDnaXorSequenceKernel <<<(cols+128-1)/128, 128 >>> (dev_dnaXor, dev_dnaProSequence, rows, cols);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dnaXorKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching dnaXorKernel!\n", cudaStatus);
    }

    //将随机dna序列与明文图像的dna进行异或操作
    dnaXorKernel <<<(rows*cols*4+128-1)/128, 128 >>> (dev_dna, dev_dnaXor, rows*cols*4);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dnaXorKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching dnaXorKernel!\n", cudaStatus);
    }

    //生成加操作序列
    unsigned char* & dev_dnaAdd = dev_dnaXor;//指针变量的引用，这样不必在gpu额外分配空间，节省时间
    getDnaAddSequenceKernel <<<(cols+128-1)/128, 128 >>> (dev_dnaAdd, dev_dnaProSequence, rows, cols);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dnaXorKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching dnaXorKernel!\n", cudaStatus);
    }

    dnaAddKernel <<<(rows*cols*4+128-1)/128, 128 >>> (dev_dna, dev_dnaAdd, rows*cols*4);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dnaXorKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching dnaXorKernel!\n", cudaStatus);
    }

    //dna解码
    dnaDecodeKernel <<<dim3((cols+128-1)/128,rows), 128 >>> (dev_dna, dev_img, dev_dnaProSequence, rows, cols);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dnaDecodeKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching dnaDecodeKernel!\n", cudaStatus);
    }
    cudaStatus=hipMemcpy(final.data, dev_img, sizeof(unsigned char) * static_cast<long long>(rows)* cols, hipMemcpyDeviceToHost);
    if(cudaStatus!= hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

    cudaStatus= hipDeviceReset();
    if(cudaStatus!= hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
    }
    //imshow("final fianl", final); waitKey(0);
    if (channels > 1) {
        //分割图像然后通道合并
        Mat* channel=new Mat[3];
        channel[0]=Mat(final,Rect(0,0,img.cols,img.rows));
        channel[1]=Mat(final,Rect(img.cols,0,img.cols,img.rows));
        channel[2]=Mat(final,Rect(img.cols*2,0,img.cols,img.rows));

        merge(channel,3,img);

        delete[] channel;
    }
    else {
        img=final.clone();
    }

}

void reSetKey() {
//sine混沌映射初始值和参数值
    std_a=3.764864654565236;//0-2*pi
    std_b=1.598741258692525;//0-2*pi
    std_k=17.589465464565456;//>0

//三维猫映射初始值和参数值
    cat3D_x = 0.192417345678913;//0-1
    cat3D_y = 0.556712345678916;//0-1
    cat3D_z = 0.932112345678123;//0-1
//洛伦兹混沌映射初始值和参数值
    lorenz_y = 0.786545641346986;//0-1
    lorenz_z = 0.253456749812345;//0-1
    lorenz_q = 0.598745498583658;//0-1

//sine2D混沌映射初始值和参数值
    sine2D_a= 37.857334516296548;//范围--负无穷到正无穷
    sine2D_b= 17.346265259595841;//范围--负无穷到正无穷

    sine2D_x= 0.265916526436985;//0-1
    sine2D_y= 0.695234546969854;//0-1

    logistic_x = 0.387368565654668;//0-1
}
