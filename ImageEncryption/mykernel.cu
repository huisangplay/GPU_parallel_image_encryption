#include "hip/hip_runtime.h"
﻿#include"mykernel.cuh"
#include<stdio.h>
__global__ void dnaDecodeKernel(unsigned char* dev_dna, unsigned char* dev_img, unsigned char* dev_dnaProSequence, int rows,int cols) {
    int j = blockIdx.x*128+threadIdx.x;
    if(j>=cols) return;
    //i行j列
    int i=blockIdx.y;
    dev_img[cols * i + j] = dnaDecode(&dev_dna[i * cols * 4 + j*4], dev_dnaProSequence[j * rows * 10 + i + rows * 9]);
}
__global__ void getDnaAddSequenceKernel(unsigned char* dev_dnaAdd, unsigned char* dev_dnaProSequence, int rows, int cols)
{
    int j = blockIdx.x*128+threadIdx.x;
    if(j>=cols) return;
    //i行j列
    for (int i = 0; i < rows; i++) {
        //这里的dna编码规则为随机生成的规则，为rows*8~rows*9
        dnaEncode(dev_dnaProSequence[j * rows * 10 + i + rows * 7], &dev_dnaAdd[i * cols * 4 + j * 4], dev_dnaProSequence[j * rows * 10 + i + rows * 8]);
    }
}

__global__ void dnaXorKernel(unsigned char* dev_dna, unsigned char* dev_dnaXor, int size) {
    int index=blockIdx.x*128+threadIdx.x;
    if(index>=size) return;
    dev_dna[index]=dnaXor(dev_dna[index],dev_dnaXor[index]);
}

__global__ void dnaAddKernel(unsigned char* dev_dna, unsigned char* dev_dnaAdd, int size)
{
    int index=blockIdx.x*128+threadIdx.x;
    if(index>=size) return;
    dev_dna[index] = dnaAdd(dev_dna[index], dev_dnaAdd[index]);
}

__global__ void getDnaXorSequenceKernel(unsigned char* dev_dnaXor, unsigned char* dev_dnaProSequence, int rows,int cols) {
    int j = blockIdx.x*128+threadIdx.x;
    if(j>=cols) return;
    //i行j列
    for (int i = 0; i < rows; i++) {
        //这里的dna编码规则为随机生成的规则，为rows*6~rows*7
        dnaEncode(dev_dnaProSequence[j * rows * 10 + i+rows*5], &dev_dnaXor[i*cols*4+j*4], dev_dnaProSequence[j * rows * 10 + i+rows*6]);
    }
}

__global__ void test(unsigned char* dev_dna, int n) {
    for (int i = n; i < 50+n; i++) {
        printf("%c",dev_dna[i]);
    }
    printf("\n");

}
__global__ void test(int* seq, int n) {
    for (int i = n; i < 50+n; i++) {
        printf("%d",seq[i]);
    }
    printf("\n");
}


__global__ void  imgDnaEncodeKernel(unsigned char* dev_img, unsigned char* dev_dna, unsigned char* dev_dnaProSequence, int rows,int cols) {
    int j = blockIdx.x*128+threadIdx.x;
    if(j>=cols) return;
    int i=blockIdx.y;
    //i行j列
    dnaEncode(dev_img[cols * i + j], &dev_dna[cols * i * 4 + j * 4], dev_dnaProSequence[j * rows * 10 + i]);
}


__global__ void getDnaProSequenceKernel(unsigned char* dev_dnaProSequence, double* dev_update, int* dev_chaos, int rows,int cols)
{
    int i = blockIdx.x*128+threadIdx.x;

    if(i>=cols) return;
    int t = 1000;

    if (dev_chaos[i] == 0) {
        double a = dev_std_a + dev_update[i];
        double b = dev_std_b+ (static_cast<double>(dev_update[i] * 1e5) - static_cast<long long>(dev_update[i] * 1e5));
        double k = dev_std_k+ (static_cast<double>(dev_update[i] * 1e10) - static_cast<long long>(dev_update[i] * 1e10));
        for (int j = 0; j < t; j++) {
            double temp_a = a + b;
            while (temp_a < 0) temp_a += 2 * dev_pi;
            temp_a = fmod(temp_a, 2 * dev_pi);
            double temp_b = b + k * sin(a + b);
            while (temp_b < 0) temp_b += 2 * dev_pi;
            temp_b = fmod(temp_b, 2 * dev_pi);

            a = temp_a;
            b = temp_b;
        }
        for (int j = 0; j < rows * 10; j++) {
            double temp_a = a + b;
            while (temp_a < 0) temp_a += 2 * dev_pi;
            temp_a = fmod(temp_a, 2 * dev_pi);
            double temp_b = b + k * sin(a + b);
            while (temp_b < 0) temp_b += 2 * dev_pi;
            temp_b = fmod(temp_b, 2 * dev_pi);

            a = temp_a;
            b = temp_b;
            long long sum = static_cast<long long>((a+b)/2 * 1e14);
            long index = rows * i * 10 + j;
            if (j < rows && j >= 0) {
                dev_dnaProSequence[index] = sum % 8;
            }
            else if (j >= rows && j < rows * 3) {
                dev_dnaProSequence[index] = sum % 4;
            }
            else if (j >= rows * 3 && j < rows * 5) {
                dev_dnaProSequence[index] = sum % 8;
            }
            else if (j >= rows * 5 && j < rows * 6) {
                dev_dnaProSequence[index] = sum % 256;
            }
            else if (j >= rows * 6 && j < rows * 7) {
                dev_dnaProSequence[index] = sum % 8;
            }
            else if (j >= rows * 7 && j < rows * 8) {
                dev_dnaProSequence[index] = sum % 256;
            }
            else if (j >= rows * 8 && j < rows * 9) {
                dev_dnaProSequence[index] = sum % 8;
            }
            else if (j >= rows * 9 && j < rows * 10) {
                dev_dnaProSequence[index] = sum % 8;
            }
        }
    }
    else if (dev_chaos[i] == 1) {
        double x = dev_cat3D_x + dev_update[i]; x = fmod(x, 1.0);
        double y = dev_cat3D_x + (static_cast<double>(dev_update[i] * 1e5) - static_cast<long long>(dev_update[i] * 1e5)); y = fmod(y, 1.0);
        double z = dev_cat3D_z + (static_cast<double>(dev_update[i] * 1e10) - static_cast<long long>(dev_update[i] * 1e10)); z = fmod(z, 1.0);

        double temp_x = x;
        double temp_y = y;
        double temp_z = z;


        for (long j = 0; j < t; ++j) {
            temp_x = 2 * x + y + 3 * z;
            temp_x = temp_x - (int)temp_x;
            temp_y = 3 * x + 2 * y + 5 * z;
            temp_y = temp_y - (int)temp_y;
            temp_z = 2 * x + y + 4 * z;
            temp_z = temp_z - (int)temp_z;

            x = temp_x;
            y = temp_y;
            z = temp_z;
        }

        for (long j = 0; j < rows * 10; ++j) {
            temp_x = 2 * x + y + 3 * z;
            temp_x = temp_x - (int)temp_x;
            temp_y = 3 * x + 2 * y + 5 * z;
            temp_y = temp_y - (int)temp_y;
            temp_z = 2 * x + y + 4 * z;
            temp_z = temp_z - (int)temp_z;

            x = temp_x;
            y = temp_y;
            z = temp_z;
            long long sum = (static_cast<long long>((x + y + z) / 3 * 1e14));
            long index = rows * i * 10 + j;
            if (j < rows && j >= 0) {
                dev_dnaProSequence[index] = sum % 8;
            }
            else if (j >= rows && j < rows * 3) {
                dev_dnaProSequence[index] = sum % 4;
            }
            else if (j >= rows * 3 && j < rows * 5) {
                dev_dnaProSequence[index] = sum % 8;
            }
            else if (j >= rows * 5 && j < rows * 6) {
                dev_dnaProSequence[index] = sum % 256;
            }
            else if (j >= rows * 6 && j < rows * 7) {
                dev_dnaProSequence[index] = sum % 8;
            }
            else if (j >= rows * 7 && j < rows * 8) {
                dev_dnaProSequence[index] = sum % 256;
            }
            else if (j >= rows * 8 && j < rows * 9) {
                dev_dnaProSequence[index] = sum % 8;
            }
            else if (j >= rows * 9 && j < rows * 10) {
                dev_dnaProSequence[index] = sum % 8;
            }

        }
    }
    else if (dev_chaos[i] == 2) {
        double y = dev_lorenz_y + dev_update[i];
        double z = dev_lorenz_z + (static_cast<double>(dev_update[i] * 1e5) - static_cast<long long>(dev_update[i] * 1e5));
        double q = dev_lorenz_q + (static_cast<double>(dev_update[i] * 1e10) - static_cast<long long>(dev_update[i] * 1e10));
        //double h = 0.005;
        double h = 0.005;
        for (int j = 0; j < t; j++) {
            double y1 = -dev_lorenz_f * y + dev_lorenz_f * z;
            double z1 = dev_lorenz_r * y - z - y * q;
            double q1 = -dev_lorenz_g * q + y * z;

            double y2 = -dev_lorenz_f * (y + h / 2 * y1) + dev_lorenz_f * (z + h / 2 * z1);
            double z2 = dev_lorenz_r * (y + h / 2 * y1) - (z + h / 2 * z1) - (y + h / 2 * y1) * (q + h / 2 * q1);
            double q2 = -dev_lorenz_g * (q + h / 2 * q1) + (y + h / 2 * y1) * (z + h / 2 * z1);

            double y3 = -dev_lorenz_f * (y + h / 2 * y2) + dev_lorenz_f * (z + h / 2 * z2);
            double z3 = dev_lorenz_r * (y + h / 2 * y2) - (z + h / 2 * z2) - (y + h / 2 * y2) * (q + h / 2 * q2);
            double q3 = -dev_lorenz_g * (q + h / 2 * q2) + (y + h / 2 * y2) * (z + h / 2 * z2);

            double y4 = -dev_lorenz_f * (y + h * y3) + dev_lorenz_f * (z + h * z3);
            double z4 = dev_lorenz_r * (y + h * y3) - (z + h * z3) - (y + h * y3) * (q + h * q3);
            double q4 = -dev_lorenz_g * (q + h * q3) + (y + h * y3) * (z + h * z3);

            double temp_y = y + h / 6 * (y1 + 2 * y2 + 2 * y3 + y4);
            double temp_z = z + h / 6 * (z1 + 2 * z2 + 2 * z3 + z4);
            double temp_q = q + h / 6 * (q1 + 2 * q2 + 2 * q3 + q4);

            y = temp_y;
            z = temp_z;
            q = temp_q;
        }
        for (int j = 0; j < rows * 10; j++) {
            double y1 = -dev_lorenz_f * y + dev_lorenz_f * z;
            double z1 = dev_lorenz_r * y - z - y * q;
            double q1 = -dev_lorenz_g * q + y * z;

            double y2 = -dev_lorenz_f * (y + h / 2 * y1) + dev_lorenz_f * (z + h / 2 * z1);
            double z2 = dev_lorenz_r * (y + h / 2 * y1) - (z + h / 2 * z1) - (y + h / 2 * y1) * (q + h / 2 * q1);
            double q2 = -dev_lorenz_g * (q + h / 2 * q1) + (y + h / 2 * y1) * (z + h / 2 * z1);

            double y3 = -dev_lorenz_f * (y + h / 2 * y2) + dev_lorenz_f * (z + h / 2 * z2);
            double z3 = dev_lorenz_r * (y + h / 2 * y2) - (z + h / 2 * z2) - (y + h / 2 * y2) * (q + h / 2 * q2);
            double q3 = -dev_lorenz_g * (q + h / 2 * q2) + (y + h / 2 * y2) * (z + h / 2 * z2);

            double y4 = -dev_lorenz_f * (y + h * y3) + dev_lorenz_f * (z + h * z3);
            double z4 = dev_lorenz_r * (y + h * y3) - (z + h * z3) - (y + h * y3) * (q + h * q3);
            double q4 = -dev_lorenz_g * (q + h * q3) + (y + h * y3) * (z + h * z3);

            double temp_y = y + h / 6 * (y1 + 2 * y2 + 2 * y3 + y4);
            double temp_z = z + h / 6 * (z1 + 2 * z2 + 2 * z3 + z4);
            double temp_q = q + h / 6 * (q1 + 2 * q2 + 2 * q3 + q4);

            y = temp_y;
            z = temp_z;
            q = temp_q;

            long long sum = static_cast<long long>(fabs(q + y + z) / 3 * 1e14);//由于洛伦兹混沌系统产生的值会有负值，所以需要求绝对值
            long index = rows * i * 10 + j;
            if (j < rows && j >= 0) {
                dev_dnaProSequence[index] = sum % 8;
            }
            else if (j >= rows && j < rows * 3) {
                dev_dnaProSequence[index] = sum % 4;
            }
            else if (j >= rows * 3 && j < rows * 5) {
                dev_dnaProSequence[index] = sum % 8;
            }
            else if (j >= rows * 5 && j < rows * 6) {
                dev_dnaProSequence[index] = sum % 256;
            }
            else if (j >= rows * 6 && j < rows * 7) {
                dev_dnaProSequence[index] = sum % 8;
            }
            else if (j >= rows * 7 && j < rows * 8) {
                dev_dnaProSequence[index] = sum % 256;
            }
            else if (j >= rows * 8 && j < rows * 9) {
                dev_dnaProSequence[index] = sum % 8;
            }
            else if (j >= rows * 9 && j < rows * 10) {
                dev_dnaProSequence[index] = sum % 8;
            }
        }
    }
}
__global__ void de_imgDnaEncodeKernel(unsigned char *dev_img, unsigned char *dev_dna, unsigned char* dev_dnaProSequence, int rows, int cols) {
    int j = blockIdx.x*128+threadIdx.x;
    if(j>=cols) return;
    //i行j列
    int i=blockIdx.y;
    dnaEncode(dev_img[cols * i + j], &dev_dna[cols * i * 4 + j * 4], dev_dnaProSequence[j * rows * 10 + i+rows*9]);
}
__global__ void dnaMinusKernel(unsigned char* dev_dna, unsigned char* dev_dnaMinus, int cols, int size)
{
    int index=blockIdx.x*128+threadIdx.x;
    if(index>=size) return;
    //dev_dna[blockIdx.x * cols + blockIdx.y] = dnaMinus(dev_dna[blockIdx.x * cols + blockIdx.y], dev_dnaMinus[blockIdx.x * cols + blockIdx.y]);
    dev_dna[index] = dnaMinus(dev_dna[index], dev_dnaMinus[index]);
}


__global__ void de_dnaDecodeKernel(unsigned char* dev_dna, unsigned char* dev_img, unsigned char* dev_dnaProSequence, int rows,int cols) {

    int j = blockIdx.x*128+threadIdx.x;
    if(j>=cols) return;
    //i行j列
    int i=blockIdx.y;
    dev_img[cols * i + j] = dnaDecode(&dev_dna[i * cols * 4 + j*4], dev_dnaProSequence[j * rows * 10 + i]);

}

__global__ void dna_sbox_replaceKernel(unsigned char* dev_dna, unsigned char* dev_dnaProSequence, int rows, int cols) {

    int j = blockIdx.x * 128 + threadIdx.x;
    if (j >= cols) return;
    int i = blockIdx.y;

    //ŒÆËãdna-sºÐÌæ»»ÐÐºÅ
    int row_index = dev_dnaProSequence[j * rows * 10 + i + rows];
    int dna_rule = dev_dnaProSequence[j * rows * 10 + i + rows * 3];
    char row_char = num_to_dna(row_index, dna_rule);

    switch (row_char) {
        case 'A':row_index = 0; break;
        case 'C':row_index = 1; break;
        case 'G':row_index = 2; break;
        case 'T':row_index = 3; break;
    }
    int col_index = dna_to_num(&dev_dna[i * cols * 2 + j * 2], dna_rule);
    dev_dna[i * cols * 2 + j * 2] = dna_sbox[dna_rule * 128 + row_index * 32 + col_index*2];
    dev_dna[i * cols * 2 + j * 2+1] = dna_sbox[dna_rule * 128 + row_index * 32 + col_index*2+1];
}
__global__ void de_dna_sbox_replaceKernel(unsigned char* dev_dna, unsigned char* dev_dnaProSequence, int rows, int cols) {
    int j = blockIdx.x * 128 + threadIdx.x;
    if (j >= cols) return;
    int i = blockIdx.y;

    //ŒÆËãdna-sºÐÌæ»»ÐÐºÅ
    int row_index = dev_dnaProSequence[j * rows * 10 + i + rows];
    int dna_rule = dev_dnaProSequence[j * rows * 10 + i + rows * 3];
    char row_char = num_to_dna(row_index, dna_rule);

    switch (row_char) {
        case 'A':row_index = 0; break;
        case 'C':row_index = 1; break;
        case 'G':row_index = 2; break;
        case 'T':row_index = 3; break;
    }

    for (int k = dna_rule * 128 + row_index * 32; k < (dna_rule * 128 + row_index * 32+32); k+=2) {
        if (dev_dna[i * cols * 2 + j * 2] == dna_sbox[k] && dev_dna[i * cols * 2 + j * 2 + 1] == dna_sbox[k + 1]) {
            int n=(k - (dna_rule * 128) - (row_index * 32))/2;
            dev_dna[i * cols * 2 + j * 2]= num_to_dna(n/4,dna_rule);
            dev_dna[i * cols * 2 + j * 2+1]= num_to_dna(n%4, dna_rule);
            break;
        }
    }
}