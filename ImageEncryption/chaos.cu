#include "hip/hip_runtime.h"
#include"chaos.cuh"
void getSine2DSequence(double* X, double* Y, int N) {
    for (int i = 0; i < N; i++) {
        double temp_x = 1 - sine2D_a * sin(sine2D_x) * sin(sine2D_x) + sine2D_y;
        temp_x = temp_x - (int)temp_x;
        double temp_y = sine2D_b * sine2D_x;
        temp_y = temp_y - (int)temp_y;
        sine2D_x = temp_x;
        sine2D_y = temp_y;

        X[i] = sine2D_x;
        Y[i] = sine2D_y;
    }
}

void getLogisticSequence(double* X, int n)
{
    for (int i = 0; i < n; i++) {
        logistic_x = logistic_u * logistic_x * (1 - logistic_x);
        X[i] = logistic_x;
    }
}
